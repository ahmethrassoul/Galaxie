
#include <hip/hip_runtime.h>
/*
 *  Copyright (c) 2022 Bertrand LE GAL
 *
 *  This software is provided 'as-is', without any express or
 *  implied warranty. In no event will the authors be held
 *  liable for any damages arising from the use of this software.
 *
 *  Permission is granted to anyone to use this software for any purpose,
 *  including commercial applications, and to alter it and redistribute
 *  it freely, subject to the following restrictions:
 *
 *  1. The origin of this software must not be misrepresented;
 *  you must not claim that you wrote the original software.
 *  If you use this software in a product, an acknowledgment
 *  in the product documentation would be appreciated but
 *  is not required.
 *
 *  2. Altered source versions must be plainly marked as such,
 *  and must not be misrepresented as being the original software.
 *
 *  3. This notice may not be removed or altered from any
 *  source distribution.
 *
 */
#if defined(__NVCC__)
#include "RenderCUDA.hpp"
#include <iostream>


RenderCUDA::RenderCUDA( struct galaxy g ) : galaxie( g )
{
    nElements = galaxie.size;

    cudaError_t err = cudaSuccess;


    c_pos = (float4*)malloc( nElements * sizeof(float4) );
    if (c_pos == nullptr) {
        fprintf(stderr, "Failed to allocate host vector c_pos !\n");
        fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    n_pos = (float4*)malloc( nElements * sizeof(float4) );
    if (n_pos == nullptr) {
        fprintf(stderr, "Failed to allocate host vector n_pos !\n");
        fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    v_dat = (float4*)malloc( nElements * sizeof(float4) );
    if (v_dat == nullptr) {
        fprintf(stderr, "Failed to allocate host vector v_dat !\n");
        fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    err = cudaMalloc((void **)&gpu_c_pos, nElements * sizeof(float4));
    if (err != cudaSuccess) {
        fprintf(stderr, "Failed to allocate device vector gpu_c_pos (error code %s)!\n", cudaGetErrorString(err));
        fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    err = cudaMalloc((void **)&gpu_n_pos, nElements * sizeof(float4));
    if (err != cudaSuccess) {
        fprintf(stderr, "Failed to allocate device vector gpu_n_pos (error code %s)!\n", cudaGetErrorString(err));
        fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    err = cudaMalloc((void **)&gpu_v_dat, nElements * sizeof(float4));
    if (err != cudaSuccess) {
        fprintf(stderr, "Failed to allocate device vector gpu_v_dat (error code %s)!\n", cudaGetErrorString(err));
        fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    //

    for(int i = 0; i < nElements; i += 1)
    {
        float4 tmp;
        tmp.x    = g.pos_x[i];
        tmp.y    = g.pos_y[i];
        tmp.z    = g.pos_z[i];
        tmp.w    = g.mass [i];
        c_pos[i] = tmp;
    }

    err = cudaMemcpy(gpu_c_pos, c_pos, nElements * sizeof(float4), cudaMemcpyHostToDevice);
    if (err != cudaSuccess)
    {
        fprintf(stderr, "Failed to copy vector c_pos from host to device (error code %s)!\n", cudaGetErrorString(err));
        fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    //

    for(int i = 0; i < nElements; i += 1)
    {
        float4 tmp;
        tmp.x        = 0.f;
        tmp.y        = 0.f;
        tmp.z        = 0.f;
        tmp.w        = 0.f;
        tmp.x    = g.pos_x[i];
        tmp.y    = g.pos_y[i];
        tmp.z    = g.pos_z[i];
        tmp.w    = g.mass [i];
        n_pos[i] = tmp;
    }

    err = cudaMemcpy(gpu_n_pos, n_pos, nElements * sizeof(float4), cudaMemcpyHostToDevice);
    if (err != cudaSuccess)
    {
        fprintf(stderr, "Failed to copy vector c_pos from host to device (error code %s)!\n", cudaGetErrorString(err));
        fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    //

    for(int i = 0; i < nElements; i += 1)
    {
        float4 tmp;
        tmp.x        = g.vel_x[i];
        tmp.y        = g.vel_y[i];
        tmp.z        = g.vel_z[i];
        tmp.w        =        0.f;
        v_dat[i] = tmp;
    }

    err = cudaMemcpy(gpu_v_dat, v_dat, nElements * sizeof(float4), cudaMemcpyHostToDevice);
    if (err != cudaSuccess)
    {
        fprintf(stderr, "Failed to copy vector v_dat from host to device (error code %s)!\n", cudaGetErrorString(err));
        fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

}
/*
 *
 *
 *##############################################################################
 *
 *
 */
RenderCUDA::~RenderCUDA()
{
    free( n_pos );
    free( c_pos );
    free( v_dat );

    cudaFree( gpu_c_pos );
    cudaFree( gpu_n_pos );
    cudaFree( gpu_v_dat );
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
extern __global__  void body2Body_exec(
                                        const float4* c_pos,
                                              float4* n_pos,
                                              float4* v_dat,
                                        const int     nElements);
//
extern __global__  void body2Body_copy(
                                              float4* c_pos,
                                        const float4* n_pos,
                                        const float4* v_dat,
                                        const int     nElements);
/*
 *
 *
 *##############################################################################
 *
 *
 */
void RenderCUDA::execute()
{
    //
    // Launch the Vector Add CUDA Kernel
    //
    cudaError_t err;
    int threadsPerBlock = 256; // max value as encoded in cuda description
    int blocksPerGrid = (nElements + threadsPerBlock - 1) / threadsPerBlock;

    body2Body_exec<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(float4)>>>(gpu_c_pos, gpu_n_pos, gpu_v_dat, nElements);

    err = cudaGetLastError();
    if (err != cudaSuccess)
    {
        fprintf(stderr, "Failed to launch body2Body_exec kernel (error code %s)!\n", cudaGetErrorString(err));
        fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    body2Body_copy<<<blocksPerGrid, threadsPerBlock>>>(gpu_c_pos, gpu_n_pos, gpu_v_dat, nElements);

    err = cudaGetLastError();
    if (err != cudaSuccess)
    {
        fprintf(stderr, "Failed to launch body2Body_copy kernel (error code %s)!\n", cudaGetErrorString(err));
        fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    //
    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    //

    err = cudaMemcpy(n_pos, gpu_n_pos, nElements * sizeof(float4), cudaMemcpyDeviceToHost);
    if (err != cudaSuccess)
    {
        fprintf(stderr, "Failed to copy vector gpu_c_pos from device to host (error code %s)!\n", cudaGetErrorString(err));
        fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    //
    // Copying particule positions to draw them in the GUI
    //

    for(int i = 0; i < nElements; i += 1)
    {
        float4 tmp = n_pos[i];
        galaxie.pos_x[i] = tmp.x;
        galaxie.pos_y[i] = tmp.y;
        galaxie.pos_z[i] = tmp.z;
    }
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
Galaxy* RenderCUDA::particules()
{
    return &galaxie;
}
//
//
//
/////////////////////////////////////////////////////////////////////////////
//
//
//
#endif