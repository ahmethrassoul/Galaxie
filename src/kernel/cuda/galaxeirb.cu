/*
 *  Copyright (c) 2022 Bertrand LE GAL
 *
 *  This software is provided 'as-is', without any express or
 *  implied warranty. In no event will the authors be held
 *  liable for any damages arising from the use of this software.
 *
 *  Permission is granted to anyone to use this software for any purpose,
 *  including commercial applications, and to alter it and redistribute
 *  it freely, subject to the following restrictions:
 *
 *  1. The origin of this software must not be misrepresented;
 *  you must not claim that you wrote the original software.
 *  If you use this software in a product, an acknowledgment
 *  in the product documentation would be appreciated but
 *  is not required.
 *
 *  2. Altered source versions must be plainly marked as such,
 *  and must not be misrepresented as being the original software.
 *
 *  3. This notice may not be removed or altered from any
 *  source distribution.
 *
 */
#include <stdio.h>
#include <hip/hip_runtime.h>
/*
 *
 *
 *##############################################################################
 *
 *
 */
#if defined(__NVCC__)
#define THREADGROUP_SIZE 256
#define BSIZE 256 			   // Block size in # of threads
#define softeningSquared 0.01f // original plumer softener is 0.025. here the value is square of it.
#define damping          1.00f // 0.999f
#define ep               0.67f // 0.5f
/*
 *
 *
 *##############################################################################
 *
 *
 */
 __global__ void body2Body_copy(
              float4* c_pos,
        const float4* n_pos,
        const float4* v_dat,
        const int   nElements)
{
    int particule_id = blockDim.x * blockIdx.x + threadIdx.x;
    
    if( particule_id < nElements )
    {
        const float4 newPos   = n_pos[ particule_id ];
        c_pos[ particule_id ] = newPos;
    }
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
__global__ void body2Body_exec(
        const float4* c_pos,
              float4* n_pos,
              float4* v_dat,
        const int     nElements)
{

    extern __shared__ float4 shPosition[THREADGROUP_SIZE];

    const int tid          = threadIdx.x;
    const int particule_id = blockIdx.x * blockDim.x + threadIdx.x;

    //
    // Si jamais a cause des threads/grilles ont deborde...
    //

    if (particule_id < nElements)
    {
        //
        // Dans le cas contraire on commence notre calcul !
        //

        float4 myPosition = c_pos[ particule_id ];
        float4 myVelocity = v_dat[ particule_id ];

        //
        //
        //

        for (int idx = 0; idx < nElements; idx += blockDim.x)
        {

            shPosition[tid] = c_pos[idx + tid];

            __syncthreads();

            #pragma unroll 4
            for (int p = 0; p < blockDim.x; p += 1)
            {

                const float4 other = shPosition[p];
                float3 r;
                r.x = other.x - myPosition.x;
                r.y = other.y - myPosition.y;
                r.z = other.z - myPosition.z;

                float d = (r.x * r.x) + (r.y * r.y) + (r.z * r.z);
                
                float d3;
                if ( d < 1.0f )
                {
                    d3 = 10.0f * other.w; // Multi modifiable
                }
                else
                {
                    d  = sqrt(d);
                    d3 = 10.0f * other.w / (d * d * d); // Multi modifiable
                }

                myVelocity.x += r.x * d3;
                myVelocity.y += r.y * d3;
                myVelocity.z += r.z * d3;
            }

            __syncthreads();

        }

        //
        // update position
        //

        __syncthreads();

        myPosition.x = myPosition.x + (myVelocity.x * 0.01f);
        myPosition.y = myPosition.y + (myVelocity.y * 0.01f);
        myPosition.z = myPosition.z + (myVelocity.z * 0.01f);

        //
        // update device memory
        //

        n_pos[ particule_id ] = myPosition;     // NEXT POSITION !
        v_dat[ particule_id ] = myVelocity;     // NEW  VELOCITY !
    }

    __syncthreads();
}

#endif